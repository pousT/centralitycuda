#include <helper_timer.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#define TIMING

/*  startTimer
 *
 *  timer - id of timer
 *
 *  Creates a new timer from the parameter.
 *  Starts the timer.
 */

void startTimer(StopWatchInterface **timer) {
    sdkCreateTimer(timer);
    checkCudaErrors( hipDeviceSynchronize() );
    sdkStartTimer(timer);
}


/*  endTimer
 *
 *  timer - id of previously started timer.
 *
 *  Waits for device to finish computing
 *  and then stops the timer.  Returns the
 *  duration of the timer.
 */

float endTimer(StopWatchInterface **timer) {

    checkCudaErrors( hipDeviceSynchronize() );
    sdkStopTimer(timer);
    float device_time = sdkGetTimerValue(timer);
    sdkDeleteTimer(timer);

    return device_time;
}
